
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

namespace detrex {
int get_cudart_version() {
  int runtimeVersion;
  hipRuntimeGetVersion(&runtimeVersion);
  return runtimeVersion;
}
} // namespace detrex
